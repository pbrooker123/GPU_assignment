#include "hip/hip_runtime.h"
﻿
//# Starting point was new cudaRuntime project in VS which was an array addition project
//# Intelisense is up and running in win10 on my alienware minitower
//# Graphics card is NVIDIA GeForce GTX 970: 4GB, 1664 cores

#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <cstdlib> // For rand() and srand()...from chatGPT
#include <ctime>   // For time()....from chatGPT

using namespace std;


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// same kernel for whether it is on GPU#1 or GPU#2
// the kernel call will pass in different array pointers
// also, the hipMemcpy() will use different source arrays
__global__ void SVKernel(int* d_forDotProduct,  int* d_SV01, int* d_SV02, int* d_SV03, int* d_SV04, int* d_SV05,
                                                int* d_SV06, int* d_SV07, int* d_SV08, int* d_SV09, int* d_SV10, 
                                                int* d_dot01, int* d_dot02, int* d_dot03, int* d_dot04, int* d_dot05, 
                                                int* d_dot06, int* d_dot07, int* d_dot08, int* d_dot09, int* d_dot10 )
{
    int i = threadIdx.x;
    d_dot01[i] = d_forDotProduct[i] * d_SV01[i];
    d_dot02[i] = d_forDotProduct[i] * d_SV02[i];
    d_dot03[i] = d_forDotProduct[i] * d_SV03[i];
    d_dot04[i] = d_forDotProduct[i] * d_SV04[i];
    d_dot05[i] = d_forDotProduct[i] * d_SV05[i];
    d_dot06[i] = d_forDotProduct[i] * d_SV06[i];
    d_dot07[i] = d_forDotProduct[i] * d_SV07[i];
    d_dot08[i] = d_forDotProduct[i] * d_SV08[i];
    d_dot09[i] = d_forDotProduct[i] * d_SV09[i];
    d_dot10[i] = d_forDotProduct[i] * d_SV10[i];
}

void DrawBoard(string boardStr[], string label)
{
    cout << endl;
    cout << " " << boardStr[0] << " | " << boardStr[1] << " | " << boardStr[2] << " | " << boardStr[3] << " " << endl;
    cout << "---|---|---|---\n";
    cout << " " << boardStr[4] << " | " << boardStr[5] << " | " << boardStr[6] << " | " << boardStr[7] << endl;
    cout << "---|---|---|---  " << label << endl;
    cout << " " << boardStr[8] << " | " << boardStr[9] << " | " << boardStr[10] << " | " << boardStr[11] << " " << endl;
    cout << "---|---|---|---\n";
    cout << " " << boardStr[12] << " | " << boardStr[13] << " | " << boardStr[14] << " | " << boardStr[15] << " " << endl;
    cout << endl;
}

string* CreateBoardString(int* h_boardOfNums)
{
    string* boardStr = new string[16];
    for (int i = 0; i < 16; ++i)
    {
        if (h_boardOfNums[i] == -1)
        {
            boardStr[i] = "X";
        } else if (h_boardOfNums[i] == 0)
        {
            boardStr[i] = "-";
        } else 
        {
            boardStr[i] = "O";
        }
    }
    return boardStr;
}


int main()
{

    cout << "**************************************\n";
    cout << "* 2 GPUs compete in 4x4 Tic-Tack-Toe *\n";
    cout << "**************************************\n\n";
    cout << endl;
    
    int nDevices;

    hipGetDeviceCount(&nDevices);

    if ( nDevices >= 1)
    {
        cout << "Number of CUDA capable devices = " << nDevices << endl;
    }
    else
    {
        cout << "No CUDA capable devices found." << endl;
        return 1;
    }


    // need to set up the 4x4=64 board
    // to pass to the device, it will just be a int* array of 0, 1 & 2
    // 0 = empty = "-"
    // 1 = X
    // 2 = O
    // initially, everything will be 0 which will be represented 

    string boardStr[16];
    for (int i = 0; i < 16; ++i)
    {
        boardStr[i] = "-";
        //cout << boardStr[i] <<" ";
    }
    
    // DrawBoard(boardStr, "Starting Board");

    // set up solution vectors for 10 possible solution. hz row, vt col, diagonal.
    int h_SV01[16] = {  1,1,1,1, 0,0,0,0, 0,0,0,0, 0,0,0,0 }; // top row
    int h_SV02[16] = {  0,0,0,0, 1,1,1,1, 0,0,0,0, 0,0,0,0 }; // 2nd row
    int h_SV03[16] = {  0,0,0,0, 0,0,0,0, 1,1,1,1, 0,0,0,0 }; // 3rd row
    int h_SV04[16] = {  0,0,0,0, 0,0,0,0, 0,0,0,0, 1,1,1,1 }; // bot row
    int h_SV05[16] = {  1,0,0,0, 1,0,0,0, 1,0,0,0, 1,0,0,0 }; // 1st col
    int h_SV06[16] = {  0,1,0,0, 0,1,0,0, 0,1,0,0, 0,1,0,0 }; // 2nd col
    int h_SV07[16] = {  0,0,1,0, 0,0,1,0, 0,0,1,0, 0,0,1,0 }; // 3rd col
    int h_SV08[16] = {  0,0,0,1, 0,0,0,1, 0,0,0,1, 0,0,0,1 }; // 4th col
    int h_SV09[16] = {  1,0,0,0, 0,1,0,0, 0,0,1,0, 0,0,0,1 }; // NW-SE diag
    int h_SV10[16] = {  0,0,0,1, 0,0,1,0, 0,1,0,0, 1,0,0,0 }; // SW-NE diag

    // GPU #1, deviceNum = 0

    //Allocate memory on GPU #1 for solution vectors

    // Loop over GPU1 & GPU2
    // If 2 GPUs, both 0 & 1 will be loaded with SVs and of course, there will be d0_ and d1_ pointers defined
    // If only 1 GPU, will still explicitly be d0_ & d1_ arrays but both will be on 0 gpu
    // If more than 2 GPUs, only 0 & 1 will be loaded with SVs

 


    // h_forDotProduct
    int* d0_forDotProduct = 0; // will differ for GPU1 vs GPU2
    //                            -1000 for others piece + 10 for your pieces, +1 for open
    hipMalloc((void**)&d0_forDotProduct, 16 * sizeof(int));
    // pointer in device for d_forDotProduct now defined on device

    int* d0_SV01 = 0;
    int* d0_SV02 = 0;
    int* d0_SV03 = 0;
    int* d0_SV04 = 0;
    int* d0_SV05 = 0;
    int* d0_SV06 = 0;
    int* d0_SV07 = 0;
    int* d0_SV08 = 0;
    int* d0_SV09 = 0;
    int* d0_SV10 = 0;
    hipMalloc((void**)&d0_SV01, 16 * sizeof(int));
    hipMalloc((void**)&d0_SV02, 16 * sizeof(int));
    hipMalloc((void**)&d0_SV03, 16 * sizeof(int));
    hipMalloc((void**)&d0_SV04, 16 * sizeof(int));
    hipMalloc((void**)&d0_SV05, 16 * sizeof(int));
    hipMalloc((void**)&d0_SV06, 16 * sizeof(int));
    hipMalloc((void**)&d0_SV07, 16 * sizeof(int));
    hipMalloc((void**)&d0_SV08, 16 * sizeof(int));
    hipMalloc((void**)&d0_SV09, 16 * sizeof(int));
    hipMalloc((void**)&d0_SV10, 16 * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(d0_SV01, h_SV01, 16 * sizeof(int), hipMemcpyHostToDevice);

    int hd0_SV01[16];
    hipMemcpy(hd0_SV01, d0_SV01, 16 * sizeof(int), hipMemcpyDeviceToHost);
    // cout << "hd0_SV01: ";
    for (int i = 0; i< 16; ++i )
    {
        // cout << hd0_SV01[i] << " ";
    }
    cout << endl;

    hipMemcpy(d0_SV02, h_SV02, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d0_SV03, h_SV03, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d0_SV04, h_SV04, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d0_SV05, h_SV05, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d0_SV06, h_SV06, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d0_SV07, h_SV07, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d0_SV08, h_SV08, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d0_SV09, h_SV09, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d0_SV10, h_SV10, 16 * sizeof(int), hipMemcpyHostToDevice);

    // need to set up memory for array multiplies the SVs with the baord array
    // cannot sum array in device. will return 10 array multiplies back and dot sum will be done on host
    int* d0_dot01 = 0;
    int* d0_dot02 = 0;
    int* d0_dot03 = 0;
    int* d0_dot04 = 0;
    int* d0_dot05 = 0;
    int* d0_dot06 = 0;
    int* d0_dot07 = 0;
    int* d0_dot08 = 0;
    int* d0_dot09 = 0;
    int* d0_dot10 = 0;
    hipMalloc((void**)&d0_dot01, 16 * sizeof(int));
    hipMalloc((void**)&d0_dot02, 16 * sizeof(int));
    hipMalloc((void**)&d0_dot03, 16 * sizeof(int));
    hipMalloc((void**)&d0_dot04, 16 * sizeof(int));
    hipMalloc((void**)&d0_dot05, 16 * sizeof(int));
    hipMalloc((void**)&d0_dot06, 16 * sizeof(int));
    hipMalloc((void**)&d0_dot07, 16 * sizeof(int));
    hipMalloc((void**)&d0_dot08, 16 * sizeof(int));
    hipMalloc((void**)&d0_dot09, 16 * sizeof(int));
    hipMalloc((void**)&d0_dot10, 16 * sizeof(int));

    // we will be explicitly defining all of the array for the 2nd GPU
    // in the case of only 1 GPU, all of the arrays will be copied onto the first GPU
    // need to change device if there are 2 GPUs and pointing to 2nd GPU
    // NOte: "1" refers to the 2nd GPU. 1st GPU is "0"
    if (nDevices > 1)
    {
        hipSetDevice(1);
    }


    // h_forDotProduct
    int* d1_forDotProduct = 0; // will differ for GPU1 vs GPU2
    //                            -1000 for others piece + 10 for your pieces, +1 for open
    hipMalloc((void**)&d1_forDotProduct, 16 * sizeof(int));
    // pointer in device for d_forDotProduct now defined on device

    int* d1_SV01 = 0;
    int* d1_SV02 = 0;
    int* d1_SV03 = 0;
    int* d1_SV04 = 0;
    int* d1_SV05 = 0;
    int* d1_SV06 = 0;
    int* d1_SV07 = 0;
    int* d1_SV08 = 0;
    int* d1_SV09 = 0;
    int* d1_SV10 = 0;
    hipMalloc((void**)&d1_SV01, 16 * sizeof(int));
    hipMalloc((void**)&d1_SV02, 16 * sizeof(int));
    hipMalloc((void**)&d1_SV03, 16 * sizeof(int));
    hipMalloc((void**)&d1_SV04, 16 * sizeof(int));
    hipMalloc((void**)&d1_SV05, 16 * sizeof(int));
    hipMalloc((void**)&d1_SV06, 16 * sizeof(int));
    hipMalloc((void**)&d1_SV07, 16 * sizeof(int));
    hipMalloc((void**)&d1_SV08, 16 * sizeof(int));
    hipMalloc((void**)&d1_SV09, 16 * sizeof(int));
    hipMalloc((void**)&d1_SV10, 16 * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(d1_SV01, h_SV01, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d1_SV02, h_SV02, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d1_SV03, h_SV03, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d1_SV04, h_SV04, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d1_SV05, h_SV05, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d1_SV06, h_SV06, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d1_SV07, h_SV07, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d1_SV08, h_SV08, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d1_SV09, h_SV09, 16 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d1_SV10, h_SV10, 16 * sizeof(int), hipMemcpyHostToDevice);

    // need to set up memory for array multiplies the SVs with the baord array
    // cannot sum array in device. will return 10 array multiplies back and dot sum will be done on host
    int* d1_dot01 = 0;
    int* d1_dot02 = 0;
    int* d1_dot03 = 0;
    int* d1_dot04 = 0;
    int* d1_dot05 = 0;
    int* d1_dot06 = 0;
    int* d1_dot07 = 0;
    int* d1_dot08 = 0;
    int* d1_dot09 = 0;
    int* d1_dot10 = 0;
    hipMalloc((void**)&d1_dot01, 16 * sizeof(int));
    hipMalloc((void**)&d1_dot02, 16 * sizeof(int));
    hipMalloc((void**)&d1_dot03, 16 * sizeof(int));
    hipMalloc((void**)&d1_dot04, 16 * sizeof(int));
    hipMalloc((void**)&d1_dot05, 16 * sizeof(int));
    hipMalloc((void**)&d1_dot06, 16 * sizeof(int));
    hipMalloc((void**)&d1_dot07, 16 * sizeof(int));
    hipMalloc((void**)&d1_dot08, 16 * sizeof(int));
    hipMalloc((void**)&d1_dot09, 16 * sizeof(int));
    hipMalloc((void**)&d1_dot10, 16 * sizeof(int));

    // the arrays for the 2 GPUs are all set
    // d0_ is 1st GPU and d1_ is the 2nd GPU


    // decide who goes first
    srand(static_cast<unsigned int>(time(0))); // Seed the random number generator

    // Generate a random number 0 or 1
    int randomNum = rand() % 2;
    // cout << "randomNum = " << randomNum << endl;

    int GPU_num;
    if (randomNum == 0)
    {
        GPU_num = -1;   // GPU1 first
        cout << "1st GPU starts";
    }
    else
    {
        GPU_num = 1; // GPU2 first
        cout << "2nd GPU starts";
    }

    int h_boardOfNums[16] = { 0 }; // this is -1 for GPU1, 0 for open, +1 for GPU2, initially, all set to 0

    // Here is where we set the initial turn
    // different start depending on if GPU1 is first or GPU2 is first
    // but no cuda calls to the kernel yet
   
    if (GPU_num == -1)
    {
        // need to udate the board with first guess at top right for GPU#1
        // int* h_boardOfNums[16] = 0; // this is -1 for GPU2, 0 for open +1 for GPU1
        h_boardOfNums[0] = -1;

    }
    else
    {
        // GPU2 only picks random space to start
        int randomNum = rand() % 15;
        h_boardOfNums[randomNum] = 1;
    }
    string* iniBoardStr = CreateBoardString(h_boardOfNums);

    cout << endl;
    DrawBoard(iniBoardStr, "End of First Turn");

    // first turn is done
    // time to switch GPU
    // since GPU_num = -1 or +1, all we need to do is to multiply by -1 to select other GPUs
    GPU_num = GPU_num * (-1);

    //cout << "New GPU_num for 2nd turn = " << GPU_num << endl;
    //cout << "h_boardOfNums: ";
    for (int i=0; i < 16; ++i)
    {
        //cout << h_boardOfNums[i] << " ";
    }
    //cout << endl;

    // here is the loop for the turns
    // will break if 4 in a row
    for (int i = 0; i < 16; ++i)
    {
        
        if (nDevices > 1 && GPU_num == 1)
        {
            hipSetDevice(1);
        }
        else
        {
            hipSetDevice(0);
        }
        // correct cuda device now set
        // will also have to use GPU_num as switch for kernel call as well as copyBack to host


        // need to create int array good for multiplying the 10 solution vectors
        // different for each GPU
        if (GPU_num == -1) 
        {
            cout << "\n\nNow 1st GPU\n";
            cout << "h_boardOfNums:           ";
            for (int i = 0; i < 16; ++i)
            {
                cout << setw(6) << h_boardOfNums[i];
            }
            cout << endl;

            cout << "Vector for SV multiply : ";
            int h_forDotProduct[16];
            for (int j =0; j < 16; ++j)
            {
                //cout << "h_boardOfNums[j] = " << h_boardOfNums[j] << endl;
                if ( h_boardOfNums[j] == -1 )
                {
                    //cout << "if true";
                    h_forDotProduct[j] = 10;
                }
                else if (h_boardOfNums[j] == 1)
                {
                    //cout << "else if true";
                    h_forDotProduct[j] = -100;
                }
                else if (h_boardOfNums[j] == 0)
                {
                    h_forDotProduct[j] = 1;
                    //cout << "else true: j= " << j << "  h_boardOfNums[j] = " << h_boardOfNums[j] << endl;
                    //cout << "forDotProduct[j] = " << forDotProduct[j] << endl;
                }
                else
                {
                    cout << "Trouble: h_BoardOfNums[j] noy -1,0, or 1" << endl;
                }
                cout << setw(6) << h_forDotProduct[j] ;
            }

            cout << endl;

            // time to copy h_forDotProduct to d_forDotProduc
            // hipMalloc((void**)&d0_forDotProduct, 16 * sizeof(int));
            hipMemcpy(d0_forDotProduct,h_forDotProduct, 16 * sizeof(int), hipMemcpyHostToDevice);

            SVKernel<<<1,16>>>(d0_forDotProduct,
                d0_SV01, d0_SV02, d0_SV03, d0_SV04, d0_SV05,
                d0_SV06, d0_SV07, d0_SV08, d0_SV09, d0_SV10,
                d0_dot01, d0_dot02, d0_dot03, d0_dot04, d0_dot05,
                d0_dot06, d0_dot07, d0_dot08, d0_dot09, d0_dot10);

            hipDeviceSynchronize();

            int hd_dot01[16], hd_dot02[16], hd_dot03[16], hd_dot04[16], hd_dot05[16], hd_dot06[16], hd_dot07[16], hd_dot08[16], hd_dot09[16], hd_dot10[16];

            hipMemcpy(hd_dot01, d0_dot01, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot02, d0_dot02, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot03, d0_dot03, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot04, d0_dot04, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot05, d0_dot05, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot06, d0_dot06, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot07, d0_dot07, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot08, d0_dot08, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot09, d0_dot09, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot10, d0_dot10, 16 * sizeof(int), hipMemcpyDeviceToHost);

            int SP01 = 0, SP02 = 0, SP03 = 0, SP04 = 0, SP05 = 0, SP06 = 0, SP07 = 0, SP08 = 0, SP09 = 0, SP10 = 0;
            int SP01_g=0, SP02_g=0, SP03_g=0, SP04_g=0, SP05_g=0, SP06_g=0, SP07_g=0, SP08_g=0, SP09_g=0, SP10_g=0; // will be index of last open in SV
            //                                                                                                      // will change if "1" is encouterred in the SV vector
            //                                                                                                      // will always return the index of the last 1 in the SV vector
            //cout << "dot01: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot01[j] << " ";
                SP01 += hd_dot01[j];
                if (hd_dot01[j] == 1)
                {
                    SP01_g = j;
                }
            }
            //cout << endl;
            //cout << "dot02: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot02[j] << " ";
                SP02 += hd_dot02[j];
                if (hd_dot02[j] == 1)
                {
                    SP02_g = j;
                }
            }
            //cout << endl;
            //cout << "dot03: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot03[j] << " ";
                SP03 += hd_dot03[j];
                if (hd_dot03[j] == 1)
                {
                    SP03_g = j;
                }
            }
            //cout << endl;
            //cout << "dot04: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot04[j] << " ";
                SP04 += hd_dot04[j];
                if (hd_dot04[j] == 1)
                {
                    SP04_g = j;
                }
            }
            //cout << endl;
            //cout << "dot05: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot05[j] << " ";
                SP05 += hd_dot05[j];
                if (hd_dot05[j] == 1)
                {
                    SP05_g = j;
                }
            }
            //cout << endl;
            //cout << "dot06: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot06[j] << " ";
                SP06 += hd_dot06[j];
                if (hd_dot06[j] == 1)
                {
                    SP06_g = j;
                }
            }
            //cout << endl;
            //cout << "dot07: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot07[j] << " ";
                SP07 += hd_dot07[j];
                if (hd_dot07[j] == 1)
                {
                    SP07_g = j;
                }
            }
            //cout << endl;
            //cout << "dot08: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot08[j] << " ";
                SP08 += hd_dot08[j];
                if (hd_dot08[j] == 1)
                {
                    SP08_g = j;
                }
            }
            //cout << endl;
            //cout << "dot09: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot09[j] << " ";
                SP09 += hd_dot09[j];
                if (hd_dot09[j] == 1)
                {
                    SP09_g = j;
                }
            }
            //cout << endl;
            //cout << "dot10: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot10[j] << " ";
                SP10 += hd_dot10[j];
                if (hd_dot10[j] == 1)
                {
                    SP10_g = j;
                }
            }
            //cout << endl;
            //cout << "SP Best guess index: " << SP01_g << " " << SP02_g << " " << SP03_g << " " << SP04_g
            //     << " " << SP05_g << " " << SP06_g << " " << SP07_g << " " << SP08_g << " " << SP09_g << " " << SP10_g << endl;

            //cout << "Scalar Product Array:" << SP01 << " " << SP02 << " " << SP03 << " " << SP04 << " " << SP05 << " " 
            //                                << SP06 << " " << SP07 << " " << SP08 << " " << SP09 << " " << SP10 << endl;
            
            int SP_values[10] = { SP01,SP02,SP03,SP04,SP05,SP06,SP07,SP08,SP09,SP10 };
            // looking for the max SP. If three are 3 X, it will be 10+10+10+1 = 31
            //  If 31 and you guess the last open place, game over!
            // Loop below is just idenifying the max Scalar Product 
            int max_SP = -1000;
            int max_SP_index = 0;
            for (int j = 0; j < 10; ++j)
            {
                if ( SP_values[j]  > max_SP )
                {
                    max_SP = SP_values[j];
                    max_SP_index = j;
                }
                // cout << "loop index: " << j << "  max_SP: " << max_SP << "  max_SP_index: " << max_SP_index << endl;
            }

            if (max_SP == 40 )
            {
                cout << "\n4 in a row for GPU1\n";
                cout << "end of game";
                return 0;
            }

            // we are on the GPU_num = -1 side. New guess needs to be set to -1. Subscript is given by SPXX_g 
            if (max_SP_index == 0)
            {
                h_boardOfNums[SP01_g] = -1;
            }
            else if (max_SP_index == 1)
            {
                h_boardOfNums[SP02_g] = -1;
            }
            else if (max_SP_index == 2)
            {
                h_boardOfNums[SP03_g] = -1;
            }
            else if (max_SP_index == 3)
            {
                h_boardOfNums[SP04_g] = -1;
            }
            else if (max_SP_index == 4)
            {
                h_boardOfNums[SP05_g] = -1;
            }
            else if (max_SP_index == 5)
            {
                h_boardOfNums[SP06_g] = -1;
            }
            else if (max_SP_index == 6)
            {
                h_boardOfNums[SP07_g] = -1;
            }
            else if (max_SP_index == 7)
            {
                h_boardOfNums[SP08_g] = -1;
            }
            else if (max_SP_index == 8)
            {
                h_boardOfNums[SP09_g] = -1;
            }
            else if (max_SP_index == 9)
            {
                h_boardOfNums[SP10_g] = -1;
            }
            else 
            {
                cout << "trouble deciding next guess\n";
            }

            // If max_SP = 31, next guess should be 4 in a row and game should end
            if (max_SP == 31)
            {

                string* iniBoardStr = CreateBoardString(h_boardOfNums);
                cout << endl;
                DrawBoard(iniBoardStr, "Four in a row for 1st GPU ");
                cout << "\n";
                cout << "\n4 in a row for GPU1\n";
                cout << "**********end of game************";
                return 0;
            }

        }
        else if (GPU_num == 1)  
        {
            cout << "\n\nNow 2nd GPU.\n";

            cout << "h_boardOfNums:           ";
            for (int i = 0; i < 16; ++i)
            {
                cout << setw(6) << h_boardOfNums[i];
            }
            cout << endl;

            cout << "Vector for SV multiply : ";
            int h_forDotProduct[16];
            for (int j = 0; j < 16; ++j)
            {
                if (h_boardOfNums[j] == -1)   //this is GPU0...1st GPU
                {
                    h_forDotProduct[j] = -100;
                }
                else if (h_boardOfNums[j] == 1)  // this is GPU1...2nd GPU
                {
                    h_forDotProduct[j] = 10;
                }
                else if (h_boardOfNums[j] == 0)
                {
                    h_forDotProduct[j] = 1;
                }
                else
                {
                    cout << "Trouble: h_BoardOfNums[j] noy -1,0, or 1" << endl;
                }
                cout << setw(6) << h_forDotProduct[j];

            }
            
            cout << endl;

            // time to copy h_forDotProduct to d_forDotProduc
            // hipMalloc((void**)&d0_forDotProduct, 16 * sizeof(int));
            hipMemcpy(d1_forDotProduct, h_forDotProduct, 16 * sizeof(int), hipMemcpyHostToDevice);

            SVKernel << <1, 16 >> > (d1_forDotProduct,
                d1_SV01,  d1_SV02,  d1_SV03,  d1_SV04,  d1_SV05,
                d1_SV06,  d1_SV07,  d1_SV08,  d1_SV09,  d1_SV10,
                d1_dot01, d1_dot02, d1_dot03, d1_dot04, d1_dot05,
                d1_dot06, d1_dot07, d1_dot08, d1_dot09, d1_dot10);


            hipDeviceSynchronize();

            int hd_dot01[16], hd_dot02[16], hd_dot03[16], hd_dot04[16], hd_dot05[16], hd_dot06[16], hd_dot07[16], hd_dot08[16], hd_dot09[16], hd_dot10[16];

            hipMemcpy(hd_dot01, d1_dot01, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot02, d1_dot02, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot03, d1_dot03, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot04, d1_dot04, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot05, d1_dot05, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot06, d1_dot06, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot07, d1_dot07, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot08, d1_dot08, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot09, d1_dot09, 16 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(hd_dot10, d1_dot10, 16 * sizeof(int), hipMemcpyDeviceToHost);

            int SP01 = 0, SP02 = 0, SP03 = 0, SP04 = 0, SP05 = 0, SP06 = 0, SP07 = 0, SP08 = 0, SP09 = 0, SP10 = 0;
            int SP01_g = 0, SP02_g = 0, SP03_g = 0, SP04_g = 0, SP05_g = 0, SP06_g = 0, SP07_g = 0, SP08_g = 0, SP09_g = 0, SP10_g = 0; // will be index of last open in SV
            //                                                                                                      // will change if "1" is encouterred in the SV vector
            //                                                                                                      // will always return the index of the last 1 in the SV vector
            //cout << "dot01: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot01[j] << " ";
                SP01 += hd_dot01[j];
                if (hd_dot01[j] == 1)
                {
                    SP01_g = j;
                }
            }
            //cout << endl;
            //cout << "dot02: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot02[j] << " ";
                SP02 += hd_dot02[j];
                if (hd_dot02[j] == 1)
                {
                    SP02_g = j;
                }
            }
            //cout << endl;
            //cout << "dot03: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot03[j] << " ";
                SP03 += hd_dot03[j];
                if (hd_dot03[j] == 1)
                {
                    SP03_g = j;
                }
            }
            //cout << endl;
            //cout << "dot04: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot04[j] << " ";
                SP04 += hd_dot04[j];
                if (hd_dot04[j] == 1)
                {
                    SP04_g = j;
                }
            }
            //cout << endl;
            //cout << "dot05: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot05[j] << " ";
                SP05 += hd_dot05[j];
                if (hd_dot05[j] == 1)
                {
                    SP05_g = j;
                }
            }
            //cout << endl;
            //cout << "dot06: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot06[j] << " ";
                SP06 += hd_dot06[j];
                if (hd_dot06[j] == 1)
                {
                    SP06_g = j;
                }
            }
            //cout << endl;
            //cout << "dot07: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot07[j] << " ";
                SP07 += hd_dot07[j];
                if (hd_dot07[j] == 1)
                {
                    SP07_g = j;
                }
            }
            //cout << endl;
            //cout << "dot08: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot08[j] << " ";
                SP08 += hd_dot08[j];
                if (hd_dot08[j] == 1)
                {
                    SP08_g = j;
                }
            }
            //cout << endl;
            //cout << "dot09: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot09[j] << " ";
                SP09 += hd_dot09[j];
                if (hd_dot09[j] == 1)
                {
                    SP09_g = j;
                }
            }
            //cout << endl;
            //cout << "dot10: ";
            for (int j = 0; j < 16; ++j)
            {
                //cout << setw(5) << hd_dot10[j] << " ";
                SP10 += hd_dot10[j];
                if (hd_dot10[j] == 1)
                {
                    SP10_g = j;
                }
            }
            //cout << endl;
            //cout << "SP Best guess index: " << SP01_g << " " << SP02_g << " " << SP03_g << " " << SP04_g
            //     << " " << SP05_g << " " << SP06_g << " " << SP07_g << " " << SP08_g << " " << SP09_g << " " << SP10_g << endl;

            //cout << "Scalar Product Array:" << SP01 << " " << SP02 << " " << SP03 << " " << SP04 << " " << SP05 << " " 
            //                                << SP06 << " " << SP07 << " " << SP08 << " " << SP09 << " " << SP10 << endl;


            //cout << "Scalar Product Array:" << SP01 << " " << SP02 << " " << SP03 << " " << SP04 << " " << SP05 << " " 
            //                                << SP06 << " " << SP07 << " " << SP08 << " " << SP09 << " " << SP10 << endl;

            int SP_values[10] = { SP01,SP02,SP03,SP04,SP05,SP06,SP07,SP08,SP09,SP10 };
            // looking for the max SP. If three are 3 X, it will be 10+10+10+1 = 31
            //  If 31 and you guess the last open place, game over!
            // Loop below is just idenifying the max Scalar Product 
            int max_SP = -1000;
            int max_SP_index = 0;
            for (int j = 0; j < 10; ++j)
            {
                if (SP_values[j] > max_SP)
                {
                    max_SP = SP_values[j];
                    max_SP_index = j;
                }
                // cout << "loop index: " << j << "  max_SP: " << max_SP << "  max_SP_index: " << max_SP_index << endl;
            }

            if (max_SP == 40)
            {
                cout << "\n4 in a row for 2nd GPU\n";
                cout << "end of game";
                return 0;
            }

            // we are on the GPU_num = +1 side. New guess needs to be set to +1. Subscript is given by SPXX_g 
            if (max_SP_index == 0)
            {
                h_boardOfNums[SP01_g] = 1;
            }
            else if (max_SP_index == 1)
            {
                h_boardOfNums[SP02_g] = 1;
            }
            else if (max_SP_index == 2)
            {
                h_boardOfNums[SP03_g] = 1;
            }
            else if (max_SP_index == 3)
            {
                h_boardOfNums[SP04_g] = 1;
            }
            else if (max_SP_index == 4)
            {
                h_boardOfNums[SP05_g] = 1;
            }
            else if (max_SP_index == 5)
            {
                h_boardOfNums[SP06_g] = 1;
            }
            else if (max_SP_index == 6)
            {
                h_boardOfNums[SP07_g] = 1;
            }
            else if (max_SP_index == 7)
            {
                h_boardOfNums[SP08_g] = 1;
            }
            else if (max_SP_index == 8)
            {
                h_boardOfNums[SP09_g] = 1;
            }
            else if (max_SP_index == 9)
            {
                h_boardOfNums[SP10_g] = 1;
            }
            else
            {
                cout << "trouble deciding next guess\n";
            }

            // If max_SP = 31, next guess should be 4 in a row and game should end
            if (max_SP == 31)
            {

                string* iniBoardStr = CreateBoardString(h_boardOfNums);
                cout << endl;
                DrawBoard(iniBoardStr, "Four in a row for 2nd GPU ");
                cout << "\n";
                cout << "\n4 in a row for 2nd GPU\n";
                cout << "**********end of game************";
                return 0;
            }

            
        }
        else
        {
            cout << "GPU_num must be -1(GPU1) or +1(GPU2)\n";
        }

        // correct  GPU set above

        // h_boardOfNums changed
        string* iniBoardStr = CreateBoardString(h_boardOfNums);

        cout << endl;
        DrawBoard(iniBoardStr, "End of Turn");

        //switch to other GPU
        GPU_num = GPU_num * (-1);
    }




    //const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };

    // Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    // }

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipDeviceReset();


    return 0;  // 0 indicates successful execution like error = 0
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
